#include "hip/hip_runtime.h"
#include <complex.h>
#include <fftw3.h>
#include <stdio.h>
#include <math.h>
#include <semaphore.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "ccs_cuda.h"

#define BLOCK_SIZE 256

// Device code
__global__ void GPU_analytic (hipfftComplex *y, unsigned int N) {
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	float da1 = 1/(float)N;
	int nh;
	
	nh = N/2+1;
	if (n > nh && n < N) {
		y[n].x = 0;
		y[n].y = 0;
	} else if (n > 0 && !(N&1)) {
		y[n].x *= 2*da1;
		y[n].y *= 2*da1;
	} else {
		y[n].x *= da1;
		y[n].y *= da1;
	}
}

__global__ void GPU_xcorr (hipfftComplex *out, hipfftComplex *xa1, hipfftComplex *xa2, float da1, unsigned int N) {
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (n<N) {
		out[n].x = da1 * (xa1[n].x * xa2[n].x + xa1[n].y * xa2[n].y);
		out[n].y = da1 * (xa1[n].y * xa2[n].x - xa1[n].x * xa2[n].y);
	}
}

__global__ void GPU_F2C (float2 *y, float *x, unsigned int N) {
	unsigned int n = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (n<N) {
		y[n].x = x[n];
		y[n].y = 0.;
	}
}

int GPU_AnalyticSignal (hipfftComplex *y, hipfftReal *x, unsigned int N, hipfftHandle *pin, hipfftHandle *pout) {
	int threadsPerBlock = BLOCK_SIZE, blocksPerGrid;

	blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	hipMemset(y, 0, N*sizeof(hipfftComplex));
	
	/* out = FFT(in) */
	if (hipfftExecR2C(*pin, x, y) != HIPFFT_SUCCESS) { fprintf(stderr, "hipfftExecR2C failed\n"); return 1; }

	/* Make it analytic ( out(w<0) = 0, out(w>0) *= 2, out(0) & out(Nyquist) no change). */
	GPU_analytic<<<blocksPerGrid, threadsPerBlock>>>(y, N);

	/* in = IFFT(out) */
	if (hipfftExecC2C(*pout, y, y, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) { fprintf(stderr, "hipfftExecC2C failed\n"); return 1; }
	
	return 0;
}

/* Sum reduction of the shared float array with the length of BLOCK_SIZE. */
template <unsigned int block_size> __device__ void SumReduction (volatile float *x, unsigned int tid) {
	if (block_size >= 1024) { if (tid < 512) x[tid] += x[tid + 512]; __syncthreads(); }
	if (block_size >= 512)  { if (tid < 256) x[tid] += x[tid + 256]; __syncthreads(); }
	if (block_size >= 256)  { if (tid < 128) x[tid] += x[tid + 128]; __syncthreads(); }
	if (block_size >= 128)  { if (tid < 64)  x[tid] += x[tid + 64];  __syncthreads(); }
	
	if (tid < 32) x[tid] += x[tid + 32];
	if (tid < 16) x[tid] += x[tid + 16];
	if (tid < 8)  x[tid] += x[tid +  8];
	if (tid < 4)  x[tid] += x[tid +  4];
	if (tid < 2)  x[tid] += x[tid +  2];
	if (tid == 0) x[0] += x[1];
}

__global__ void GPU_norm_ln(float *y, float2 *x, unsigned int N) {
	__shared__ float a[BLOCK_SIZE];
	
	unsigned int tid = threadIdx.x;
	unsigned int i = 2*blockIdx.x*blockDim.x + tid;
	unsigned int j;
	
	a[tid] = 0;
	if (i<N) a[tid]  = x[i].x*x[i].x + x[i].y*x[i].y; 
	__syncthreads();
	
	j = i + blockDim.x;
	if (j<N) a[tid] += x[j].x*x[j].x + x[j].y*x[j].y;
	__syncthreads();
	
	SumReduction <BLOCK_SIZE> (a, tid);
	
	if (tid == 0) y[blockIdx.x] = a[0]; /* Here each block has the squared norm of its elements */
}

__global__ void GPU_mabs_reduction(float *x, unsigned int M, unsigned int N) {
	__shared__ float a[BLOCK_SIZE];
	
	unsigned int tid = threadIdx.x;
	unsigned int i = 2*blockIdx.x*blockDim.x + tid;
	
	a[tid] = 0;
	if (i<M) a[tid] = x[i];
	__syncthreads();
	
	if (i + blockDim.x<M) a[tid] += x[i + blockDim.x];
	__syncthreads();
	
	SumReduction <BLOCK_SIZE> (a, tid);
	
	if (tid == 0) {
		if (blockDim.x == 1) x[0] = sqrtf(a[0])/N;
		x[blockIdx.x] = a[0];
	}
}

__global__ void GPU_AmpNormf2(float2 *x, unsigned int N, float *mabs) {   /* Add eps */
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	
	float e, fa1, fa2, fa3;
	
	if (n<N) {
		e = 1e-6*mabs[0];
		fa1 = x[n].x;
		fa2 = x[n].y;
		fa3 = fa1*fa1+fa2*fa2;
		if (fa3 != 0) {
			fa3 = 1/(sqrtf(fa3) + e);
			x[n].x = fa1*fa3;
			x[n].y = fa2*fa3;
		} else {
			x[n].x = 0;
			x[n].y = 0;
		}
	}
}

__global__ void GPU_AmpNormf(float2 *x, unsigned int N, float mabs) {   /* Add eps */
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	
	float e, fa1, fa2, fa3;
	
	if (n<N) {
		e = 1e-6*mabs;
		fa1 = x[n].x;
		fa2 = x[n].y;
		fa3 = fa1*fa1+fa2*fa2;
		if (fa3 != 0) {
			fa3 = 1/(sqrtf(fa3) + e);
			x[n].x = fa1*fa3;
			x[n].y = fa2*fa3;
		} else {
			x[n].x = 0;
			x[n].y = 0;
		}
	}
}

__global__ void PCC1_lowlevel (float *x1, float *x2, float *y, int N, const int L, const int l1, const int Lag1) {
	int l = blockDim.x * blockIdx.x + threadIdx.x;
	
	float fa1, fa2, fa3, fa4, fa5;
	float xa[2], xb[2];
	int lag, n;
	
	if (l >= l1 && l < L) {
		lag = Lag1 + l;
		if (lag < 0) { x2 -= 2*lag;  N += lag; }
		else         { x1 += 2*lag;  N -= lag; }
		
		fa5 = 0;
		for (n=0; n<2*N; n+=2) {
			xa[0] = x1[n]; xa[1] = x1[n+1];
			xb[0] = x2[n]; xb[1] = x2[n+1];
			fa1 = xa[0] + xb[0];
			fa2 = xa[1] + xb[1];
			fa3 = xa[0] - xb[0];
			fa4 = xa[1] - xb[1];
			fa5 += sqrtf(fa1*fa1+fa2*fa2) - sqrtf(fa3*fa3+fa4*fa4);
		}
		y[l] = fa5/(2*N);
	}
}

__global__ void PCC1_lowlevel2 (const float2 *x1, const float2 *x2, float *y, int N, const int L, const int l1, const int Lag1) {
	int l = blockDim.x * blockIdx.x + threadIdx.x; /* One thread per lag */
	int ls = threadIdx.x;
	__shared__ float2 x1s[2*BLOCK_SIZE]; /* x1 & x2 are in fact float-complex types */
	__shared__ float2 x2s[BLOCK_SIZE];

	float fa1, fa2, fa3, fa4, fa5;
	float2 *xa, *xb;
	int n, nb;
	
	/* The smallest domain is one block */
	fa5 = 0;
	for (nb=0; nb<N; nb+=BLOCK_SIZE) {
		/*** Copy data from the device to the shared memory ***/ 
		n = nb + ls; /* One sample of x1[n] */
		if (n < N) {
			x2s[ls].x = x2[n].x;
			x2s[ls].y = x2[n].y;
		} else {
			x2s[ls].x = 0;
			x2s[ls].y = 0;
		}
		
		n = nb + Lag1 + l; /* Sample from x2 aligned to the sample of x1 read above when threadIdx.x = 0. */
		if (n >= 0 && n < N) {
			x1s[ls].x = x1[n].x;
			x1s[ls].y = x1[n].y;
		} else {
			x1s[ls].x = 0;
			x1s[ls].y = 0;
		}
		
		n  += BLOCK_SIZE;
		ls += BLOCK_SIZE;
		if (n >= 0 && n < N) {
			x1s[ls].x = x1[n].x;
			x1s[ls].y = x1[n].y;
		} else {
			x1s[ls].x = 0;
			x1s[ls].y = 0;
		}
		
		__syncthreads();
		
		/*** Partial PCC1 ***/
		ls = threadIdx.x;
		if (l >= l1 && l < L) {
			xb = x1s + ls;
			xa = x2s;
			for (n=0; n<BLOCK_SIZE; n++) {
				fa1 = xa[n].x + xb[n].x;
				fa2 = xa[n].y + xb[n].y;
				fa3 = xa[n].x - xb[n].x;
				fa4 = xa[n].y - xb[n].y;
				fa5 += sqrtf(fa1*fa1+fa2*fa2) - sqrtf(fa3*fa3+fa4*fa4);
			}
		}
		__syncthreads();
	}
	if (l >= l1 && l < L)
		y[l] = fa5/(2*(N-abs(Lag1 + l)));
}

// Host code

void cuda_AmpNormf(float2 *x, unsigned int N) {
	float eps, *norm_ln;
	int M, nth = BLOCK_SIZE, nbl;
	// loat *tmp;
	// int m;
	
	/* Compute reguralitzation as 1e-6 part of the mean modulus of x. */
	nbl = (N + 2*nth - 1) / (2*nth);                     /* One thread per two elements on x. */
	hipMalloc(&norm_ln, nbl*sizeof(float));             /* Buffer for partiar norms.         */
	// hipHostMalloc(&tmp, nbl*sizeof(float));             /* Buffer for partiar norms.         */
	
	GPU_norm_ln<<<nbl, nth>>>(norm_ln, x, N);  /* First iteration, squared modulus of each block */
	// hipMemcpy(tmp, norm_ln, nbl*sizeof(float), hipMemcpyDeviceToHost);
	
	/* printf("\nM=%d\n", nbl);
	for (m=0; m<nbl; m++) printf("%g ", tmp[m]);
	printf("\n"); */
	
	while (nbl > 1) {
		M = nbl;
		nbl = (M + 2*nth - 1) / (2*nth);
		// printf("nbl=%d\n", nbl);
		GPU_mabs_reduction<<<nbl, nth>>>(norm_ln, M, N); /* Iterate until a single block remains. */
	}
	hipMemcpy(&eps, norm_ln, sizeof(float), hipMemcpyDeviceToHost);
	
	// printf("N=%d Norm^2=%g\n", N, eps);
	hipFree(norm_ln);
	// hipHostFree(tmp);
	
	/* Amplitude Normalization */
	nbl = (N + nth - 1) / nth;
	// printf("eps = %g\n", eps);
	GPU_AmpNormf<<<nbl, nth>>>(x, N, eps);
}

void cuda_AmpNormf2(float2 *x, unsigned int N, float *norm_ln, hipStream_t *stream) {
	int M, nth = BLOCK_SIZE, nbl;
	
	/* Compute reguralitzation as 1e-6 part of the mean modulus of x. */
	nbl = (N + 2*nth - 1) / (2*nth);                     /* One thread per two elements on x. */
	if (stream) GPU_norm_ln<<<nbl, nth, 0, stream[0]>>>(norm_ln, x, N);  /* First iteration, squared modulus of each block */
	else		GPU_norm_ln<<<nbl, nth>>>(norm_ln, x, N); 
	while (nbl > 1) {
		M = nbl;
		nbl = (M + 2*nth - 1) / (2*nth);
		if (stream) GPU_mabs_reduction<<<nbl, nth, 0, stream[0]>>>(norm_ln, M, N); /* Iterate until a single block remains. */
		else        GPU_mabs_reduction<<<nbl, nth>>>(norm_ln, M, N); /* Iterate until a single block remains. */
	}
	
	/* Amplitude Normalization */
	nbl = (N + nth - 1) / nth;
	if (stream)  GPU_AmpNormf2<<<nbl, nth, 0, stream[0]>>>(x, N, norm_ln);
	else         GPU_AmpNormf2<<<nbl, nth>>>(x, N, norm_ln);
}

int pcc1_highlevel_error (const char *str, hipError_t cudaerr) {
	printf ("%s :%s\n", str, hipGetErrorString(cudaerr));
	hipDeviceReset();
	return -1;
}

#if 0
int pcc1_highlevel (float **y, _Complex float **x1, _Complex float **x2, int N, unsigned int Tr, int Lag1, int Lag2, sem_t *anok) {
	float *h_y;
	float2 *d_xan1, *d_xan2;
	float *d_y;
	size_t szx, szy;
	unsigned int tr, l, l1, L=(unsigned)abs(Lag2-Lag1+1);
	int threadsPerBlock = BLOCK_SIZE, blocksPerGrid;
	
	if (Lag2 > N) L -= (Lag2-N);
	l1 = (Lag1 >= -N) ? 0 : -(Lag1+N);

	/* Convert data to float */
	szx = N*sizeof(float2);
	szy = L*sizeof(float);
	
	/* Allocate vectors in device & host memories */
	hipHostMalloc(&h_y, szy);
	hipMalloc(&d_xan1, szx);
	hipMalloc(&d_xan2, szx);
	hipMalloc(&d_y,    szy);
	
	for (tr=0; tr<Tr; tr++) {
		sem_wait(&anok[tr]);
		
		hipMemcpy(d_xan1, (float *)x1[tr], szx, hipMemcpyHostToDevice);
		hipMemcpy(d_xan2, (float *)x2[tr], szx, hipMemcpyHostToDevice);
		
		// Invoke the kernel GPU_AmpNormf including regurization.
		cuda_AmpNormf(d_xan1, N);
		cuda_AmpNormf(d_xan2, N);
		
		// Invoke the kernel PCC1_lowlevel
		blocksPerGrid = (L-l1 + threadsPerBlock - 1) / threadsPerBlock;
		PCC1_lowlevel2<<<blocksPerGrid, threadsPerBlock>>>(d_xan1, d_xan2, d_y, N, L, l1, Lag1);
		
		/* Copy result back */
		hipMemcpy(h_y, d_y, szy, hipMemcpyDeviceToHost);
		for (l=l1; l<L; l++) y[tr][l] = h_y[l];
	}
	
	hipFree(d_xan1);
	hipFree(d_xan2);
	hipFree(d_y);
	hipHostFree(h_y);
	
	return 0;
}
#else
int pcc1_highlevel (float **y, _Complex float **x1, _Complex float **x2, int N, unsigned int Tr, int Lag1, int Lag2, sem_t *anok) {
	float *h_y;
	float2 *d_xan1, *d_xan2;
	float *d_y;
	size_t szx, szy;
	unsigned int tr, l, l1, n, m, L=(unsigned)abs(Lag2-Lag1+1);
	int threadsPerBlock = BLOCK_SIZE, blocksPerGrid1, blocksPerGrid2;
	size_t available=0, total=0;
	hipStream_t stream[16];
	hipError_t cudaerr;
	
	if (Lag2 > N) L -= (Lag2-N);
	l1 = (Lag1 >= -N) ? 0 : -(Lag1+N);

	/* Convert data to float */
	szx = N*sizeof(float2);
	szy = L*sizeof(float);
	
	cudaerr = hipMemGetInfo(&available, &total);
	if (cudaerr != hipSuccess) 
		{ printf ("Error getting memory info (%s)\n", hipGetErrorString(cudaerr)); return -1; }
	
	/* Allocate vectors in device & host memories */
	cudaerr = hipHostMalloc(&h_y, Tr*szy);
		if (cudaerr != hipSuccess) return pcc1_highlevel_error("Error allocating pinned host memory h_y", cudaerr);
	hipMalloc(&d_xan1, 16*szx);
	hipMalloc(&d_xan2, 16*szx);
	hipMalloc(&d_y,    16*szy);
	
	blocksPerGrid1 = (N + threadsPerBlock - 1) / threadsPerBlock;
	blocksPerGrid2 = (L-l1 + threadsPerBlock - 1) / threadsPerBlock;
	for (m=0; m<16; m++) hipStreamCreate(&stream[m]);
	for (n=0; n<((unsigned)Tr+15)/16; n++) {
		for (m=0; m<16; m++) {
			tr = 16*n+m;
			if (tr < (unsigned)Tr) {
				sem_wait(&anok[tr]);
				
				// Copy data to the GPU memory
				hipMemcpyAsync(d_xan1 + m*N, (float *)x1[tr], szx, hipMemcpyHostToDevice, stream[m]);
				hipMemcpyAsync(d_xan2 + m*N, (float *)x2[tr], szx, hipMemcpyHostToDevice, stream[m]);
				
				// Invoke the kernel GPU_AmpNormf
				GPU_AmpNormf<<<blocksPerGrid1, threadsPerBlock, 0, stream[m]>>>(d_xan1 + m*N, N, 0);
				GPU_AmpNormf<<<blocksPerGrid1, threadsPerBlock, 0, stream[m]>>>(d_xan2 + m*N, N, 0);
				
				// Invoke the kernel PCC1_lowlevel
				PCC1_lowlevel2<<<blocksPerGrid2, threadsPerBlock, 0, stream[m]>>>(d_xan1 + m*N, d_xan2 + m*N, d_y + m*L, N, L, l1, Lag1);
				
				/* Copy result back */
				hipMemcpyAsync(h_y + tr*L, d_y + m*L, szy, hipMemcpyDeviceToHost, stream[m]);
			}
		}
	}
	hipDeviceSynchronize();
	
	for (m=0; m<16; m++) hipStreamDestroy(stream[m]);
	
	for (tr=0; tr<Tr; tr++)
		for (l=l1; l<L; l++) y[tr][l] = h_y[tr*L + l];
	
	hipFree(d_xan1);
	hipFree(d_xan2);
	hipFree(d_y);
	hipHostFree(h_y);
	
	return 0;
}
#endif

void pcc1_highlevel2 (float **y, float **x1, float **x2, int N, unsigned int Tr, int Lag1, int Lag2) {
	hipfftHandle pin, pout;
	hipfftReal *d_x=NULL;
	float *h_x, *h_y, *d_y, *mem;
	float2 *d_xan1, *d_xan2;
	size_t szxr, szxc, szy;
	unsigned int tr, l, l1, L=(unsigned)abs(Lag2-Lag1+1);
	int threadsPerBlock = BLOCK_SIZE, blocksPerGrid;
	
	blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	
	if (Lag2 > N) L -= (Lag2-N);
	l1 = (Lag1 >= -N) ? 0 : -(Lag1+N);

	/* Convert data to float */
	szxr = N*sizeof(float);
	szxc = N*sizeof(float2);
	szy = L*sizeof(float);
	
	hipHostMalloc(&h_x, szxr);
	hipHostMalloc(&h_y, szy);
	
	/* Allocate vectors in device memory */
	hipMalloc(&d_x,    szxr);
	hipMalloc(&d_xan1, szxc);
	hipMalloc(&d_xan2, szxc);
	hipMalloc(&d_y,    szy);
	hipMalloc(&mem, blocksPerGrid*sizeof(float));
	
	/* Make the plans */
	if (hipfftPlan1d(&pin,  N, HIPFFT_R2C, 1) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFPlan1d failed\n"); return; }
	if (hipfftPlan1d(&pout, N, HIPFFT_C2C, 1) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFPlan1d failed\n"); return; }
	
	for (tr=0; tr<Tr; tr++) {
		memcpy(h_x, x1[tr], N*sizeof(float));
		hipMemcpy(d_x, h_x, szxr, hipMemcpyHostToDevice);
		GPU_AnalyticSignal ((hipfftComplex *)d_xan1, d_x, N, &pin, &pout);
		
		memcpy(h_x, x2[tr], N*sizeof(float));
		hipMemcpy(d_x, h_x, szxr, hipMemcpyHostToDevice);
		GPU_AnalyticSignal ((hipfftComplex *)d_xan2, d_x, N, &pin, &pout);
		
		// Invoke the kernel GPU_AmpNormf
		cuda_AmpNormf2(d_xan1, N, mem, NULL);
		cuda_AmpNormf2(d_xan2, N, mem, NULL);
		
		// Invoke the kernel PCC1_lowlevel
		blocksPerGrid = (L-l1 + threadsPerBlock - 1) / threadsPerBlock;
		PCC1_lowlevel2<<<blocksPerGrid, threadsPerBlock>>>(d_xan1, d_xan2, d_y, N, L, l1, Lag1);
		
		/* Copy result back */
		hipMemcpy(h_y, d_y, szy, hipMemcpyDeviceToHost);
		for (l=l1; l<L; l++) y[tr][l] = h_y[l];
	}
	
	/* Destroy the plans */
	hipfftDestroy(pin);
	hipfftDestroy(pout);
	
	hipFree(d_xan1);
	hipFree(d_xan2);
	hipFree(d_y);
	hipFree(d_x);
	
	hipHostFree(h_y);
	hipHostFree(h_x);
}
